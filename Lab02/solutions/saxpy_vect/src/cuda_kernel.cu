#include "hip/hip_runtime.h"
// CUDA libraries.
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

// Include associated header file.
#include "../include/cuda_kernel.cuh"


/**
 * Sample CUDA device function which adds an element from array A and array B.
 *
 */
__global__ void VectorAdd(float *A, float *B, float *C){
   int tid = blockDim.x*blockIdx.x + threadIdx.x;
   C[tid] += A[tid]*B[tid];
}



/**
 * Wrapper function for the CUDA kernel function.
 */
void kernel(float *A, float *B, float *C, int N) {
    // Launch CUDA kernel.
    float *d_A, *d_B, *d_C;
    hipMalloc((void**) &d_A, N*sizeof(float));
    hipMalloc((void**) &d_B, N*sizeof(float));
    hipMalloc((void**) &d_C, N*sizeof(float));

    hipMemcpy(d_A, A, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, N*sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(512,1,1);
    dim3 gridSize(N/512+1,1,1);

    VectorAdd<<<gridSize, blockSize>>>(d_A, d_B, d_C);
    
    hipMemcpy(C, d_C, N*sizeof(float), hipMemcpyDeviceToHost);
}











