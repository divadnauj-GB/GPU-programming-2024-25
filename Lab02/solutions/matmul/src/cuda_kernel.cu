#include "hip/hip_runtime.h"
// CUDA libraries.
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

// Include associated header file.
#include "../include/cuda_kernel.cuh"





/**
 * Sample CUDA device function which adds an element from array A and array B.
 *
 */
__global__ void MatMul(float *A, float *B, float *C, int M, int N, int K){
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    if (row<M && col<N){
        for (int ii = 0; ii < K; ii++) {
            C[row * N + col] += A[row * K + ii] * B[ii * N + col];
        }
    }
}



/**
 * Wrapper function for the CUDA kernel function.
 */
void kernel(float *A, float *B, float *C, int M, int N, int K) {
    // Launch CUDA kernel.
    float *d_A, *d_B, *d_C;
    hipMalloc((void**) &d_A, M*K*sizeof(float));
    hipMalloc((void**) &d_B, K*N*sizeof(float));
    hipMalloc((void**) &d_C, M*N*sizeof(float));

    hipMemcpy(d_A, A, M*K*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, K*N*sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(32,32);
    dim3 gridSize(ceil(((float)N)/blockSize.x), 
                  ceil(((float)M)/blockSize.y));

    MatMul<<<gridSize, blockSize>>>(d_A, d_B, d_C, M, N, K);
    //hipDeviceSynchronize();
    hipMemcpy(C, d_C, M*N*sizeof(float), hipMemcpyDeviceToHost);
}











